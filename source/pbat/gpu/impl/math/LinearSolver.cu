// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "LinearSolver.cuh"

namespace pbat::gpu::impl::math {

LinearSolver::LinearSolver() : mIsInitialized(false)
{
    hipsolverStatus_t status = hipsolverDnCreate(&mCusolverHandle);
    mIsInitialized          = (status == hipsolverStatus_t::HIPSOLVER_STATUS_SUCCESS);
}

bool LinearSolver::IsInitialized() const
{
    return mIsInitialized;
}

LinearSolver::~LinearSolver()
{
    if (mIsInitialized)
    {
        hipsolverDnDestroy(mCusolverHandle);
    }
}

} // namespace pbat::gpu::impl::math
