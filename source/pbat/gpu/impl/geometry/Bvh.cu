#include "hip/hip_runtime.h"
// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "Bvh.cuh"
#include "pbat/HostDevice.h"

#include <cuda/atomic>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <type_traits>

namespace pbat {
namespace gpu {
namespace impl {
namespace geometry {
namespace kernels {

namespace mini = pbat::math::linalg::mini;

struct FGenerateHierarchy
{
    using MortonCodeType = typename Bvh::MortonCodeType;

    struct Range
    {
        GpuIndex i, j, l;
        int d;
    };

    PBAT_DEVICE int Delta(GpuIndex i, GpuIndex j) const
    {
        if (j < 0 or j >= n)
            return -1;
        if (morton[i] == morton[j])
            return sizeof(MortonCodeType) * 8 /* #bits/byte */ + __clz(i ^ j);
        return __clz(morton[i] ^ morton[j]);
    }

    PBAT_DEVICE Range DetermineRange(GpuIndex i) const
    {
        // Compute range direction
        bool const dsign = (Delta(i, i + 1) - Delta(i, i - 1)) > 0;
        int const d      = 2 * dsign - 1;
        // Lower bound on length of internal node i's common prefix
        int const dmin = Delta(i, i - d);
        // Compute conservative upper bound on the range's size
        GpuIndex lmax{2};
        while (Delta(i, i + lmax * d) > dmin)
            lmax <<= 1;
        // Binary search in the "inflated" range for the actual end (or start) of internal node i's
        // range, considering that i is its start (or end).
        GpuIndex l{0};
        do
        {
            lmax >>= 1;
            if (Delta(i, i + (l + lmax) * d) > dmin)
                l += lmax;
        } while (lmax > 1);
        GpuIndex j = i + l * d;
        return Range{i, j, l, d};
    }

    PBAT_DEVICE GpuIndex FindSplit(Range R) const
    {
        // Calculate the number of highest bits that are the same
        // for all objects.
        int const dnode = Delta(R.i, R.j);

        // Use binary search to find where the next bit differs.
        // Specifically, we are looking for the highest object that
        // shares more than dnode bits with the first one.
        GpuIndex s{0};
        do
        {
            R.l = (R.l + 1) >> 1;
            if (Delta(R.i, R.i + (s + R.l) * R.d) > dnode)
                s += R.l;
        } while (R.l > 1);
        GpuIndex gamma = R.i + s * R.d + min(R.d, 0);
        return gamma;
    }

    PBAT_DEVICE void operator()(auto in)
    {
        // Find out which range of objects the node corresponds to.
        Range R = DetermineRange(in);
        // Determine where to split the range.
        GpuIndex gamma = FindSplit(R);
        // Select left+right child
        GpuIndex i  = min(R.i, R.j);
        GpuIndex j  = max(R.i, R.j);
        GpuIndex lc = (i == gamma) ? leafBegin + gamma : gamma;
        GpuIndex rc = (j == gamma + 1) ? leafBegin + gamma + 1 : gamma + 1;
        // Record parent-child relationships
        child[0][in] = lc;
        child[1][in] = rc;
        parent[lc]   = in;
        parent[rc]   = in;
        // Record subtree relationships
        rightmost[0][in] = leafBegin + gamma;
        rightmost[1][in] = leafBegin + j;
    }

    MortonCodeType const* morton;
    std::array<GpuIndex*, 2> child;
    GpuIndex* parent;
    std::array<GpuIndex*, 2> rightmost;
    GpuIndex leafBegin;
    GpuIndex n;
};

} // namespace kernels

Bvh::Bvh(GpuIndex nBoxes)
    : inds(nBoxes),
      morton(nBoxes),
      child(nBoxes - 1),
      parent(2 * nBoxes - 1),
      rightmost(nBoxes - 1),
      iaabbs(nBoxes - 1),
      visits(nBoxes - 1)
{
    parent.SetConstant(GpuIndex(-1));
}

void Bvh::Build(Aabb<kDims>& aabbs, Morton::Bound const& WL, Morton::Bound const& WU)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(ctx, "pbat.gpu.impl.geometry.Bvh.Build");
    GpuIndex const n = aabbs.Size();
    SortByMortonCode(aabbs, WL, WU);
    BuildTree(n);
    ConstructBoxes(aabbs);
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(ctx);
}

void Bvh::SortByMortonCode(Aabb<kDims>& aabbs, Morton::Bound const& WL, Morton::Bound const& WU)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(
        sortCtx,
        "pbat.gpu.impl.geometry.Bvh.SortByMortonCode");

    auto const n = aabbs.Size();
    morton.Encode(aabbs, WL, WU);
    thrust::sequence(thrust::device, inds.Data(), inds.Data() + n);
    auto zip = thrust::make_zip_iterator(
        aabbs.b[0].begin(),
        aabbs.b[1].begin(),
        aabbs.b[2].begin(),
        aabbs.e[0].begin(),
        aabbs.e[1].begin(),
        aabbs.e[2].begin(),
        inds.Data());
    // Using a stable sort preserves the initial ordering of simplex indices 0...n-1, resulting in
    // simplices sorted by Morton codes first, and then by simplex index.
    thrust::stable_sort_by_key(thrust::device, morton.codes.Data(), morton.codes.Data() + n, zip);

    PBAT_PROFILE_CUDA_HOST_SCOPE_END(sortCtx);
}

void Bvh::BuildTree(GpuIndex n)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(hierarchyCtx, "pbat.gpu.impl.geometry.Bvh.BuildTree");
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(n - 1),
        kernels::FGenerateHierarchy{
            morton.codes.Raw(),
            child.Raw(),
            parent.Raw(),
            rightmost.Raw(),
            n - 1,
            n});
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(hierarchyCtx);
}

void Bvh::ConstructBoxes(Aabb<kDims>& aabbs)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(
        iaabbCtx,
        "pbat.gpu.impl.geometry.Bvh.Build.InternalAabbs");
    visits.SetConstant(GpuIndex(0));
    auto const n = aabbs.Size();
    auto& b      = aabbs.b;
    auto& e      = aabbs.e;
    auto& ib     = iaabbs.b;
    auto& ie     = iaabbs.e;
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(n - 1),
        thrust::make_counting_iterator(2 * n - 1),
        [leafBegin = n - 1,
         parent    = parent.Raw(),
         child     = child.Raw(),
         b         = b.Raw(),
         e         = e.Raw(),
         ib        = ib.Raw(),
         ie        = ie.Raw(),
         visits    = visits.Raw()] PBAT_DEVICE(auto leaf) {
            auto p = parent[leaf];
            auto k = 0;
            for (; (k < 64) and (p >= 0); ++k)
            {
                cuda::atomic_ref<GpuIndex, cuda::thread_scope_device> ap{visits[p]};
                // The first thread that gets access to the internal node p will terminate,
                // while the second thread visiting p will be allowed to continue execution.
                // This ensures that there is no race condition where a thread can access an
                // internal node too early, i.e. before both children of the internal node
                // have finished computing their bounding boxes.
                if (ap++ == 0)
                    break;

                GpuIndex lc             = child[0][p];
                GpuIndex rc             = child[1][p];
                bool const bIsLeftLeaf  = lc >= leafBegin;
                bool const bIsRightLeaf = rc >= leafBegin;
                lc -= bIsLeftLeaf * leafBegin;
                rc -= bIsRightLeaf * leafBegin;
                auto bl = bIsLeftLeaf ? b : ib;
                auto el = bIsLeftLeaf ? e : ie;
                auto br = bIsRightLeaf ? b : ib;
                auto er = bIsRightLeaf ? e : ie;
                for (auto d = 0; d < 3; ++d)
                {
                    ib[d][p] = min(bl[d][lc], br[d][rc]);
                    ie[d][p] = max(el[d][lc], er[d][rc]);
                }
                // Move up the binary tree
                p = parent[p];
            }
            assert(k < 64);
        });
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(iaabbCtx);
}

} // namespace geometry
} // namespace impl
} // namespace gpu
} // namespace pbat

#include "pbat/common/ConstexprFor.h"
#include "pbat/common/Eigen.h"
#include "pbat/geometry/DistanceQueries.h"
#include "pbat/gpu/impl/common/SynchronizedList.cuh"

#include <algorithm>
#include <cuda/std/utility>
#include <doctest/doctest.h>
#include <unordered_set>

#pragma nv_diag_suppress 177

namespace pbat {
namespace gpu {
namespace impl {
namespace geometry {
namespace test {
namespace Bvh {

struct FOnOverlapDetected
{
    using Overlap = cuda::std::pair<GpuIndex, GpuIndex>;
    std::array<GpuIndex*, 4> T;
    common::DeviceSynchronizedList<Overlap> o;
    PBAT_DEVICE void operator()(GpuIndex si, GpuIndex sj)
    {
        using namespace pbat::math::linalg::mini;
        auto ti = FromBuffers<4, 1>(T, si);
        auto tj = FromBuffers<4, 1>(T, sj);
        bool bConnected{false};
        pbat::common::ForRange<0, 4>([&]<auto d>() { bConnected |= Any(tj == ti[d]); });
        if (not bConnected)
        {
            o.Append(Overlap{si, sj});
        }
    };
};

using TQuery = pbat::math::linalg::mini::SVector<GpuScalar, 3>;
using TLeaf  = pbat::math::linalg::mini::SMatrix<GpuScalar, 3, 4>;
using TPoint = pbat::math::linalg::mini::SVector<GpuScalar, 3>;
using math::linalg::mini::FromBuffers;

namespace DistanceQueries = pbat::geometry::DistanceQueries;

struct FGetQueryObject
{
    std::array<GpuScalar*, 3> pts;
    PBAT_DEVICE TQuery operator()(GpuIndex q) const { return FromBuffers<3, 1>(pts, q); }
};

struct FDistancePointAabb
{
    PBAT_DEVICE GpuScalar operator()(TQuery const& Q, TPoint const& L, TPoint const& U) const
    {
        return DistanceQueries::PointAxisAlignedBoundingBox(Q, L, U);
    }
};

struct FDistancePointTetrahedron
{
    std::array<GpuScalar*, 3> verts;
    std::array<GpuIndex*, 4> tets;
    PBAT_DEVICE GpuScalar
    operator()([[maybe_unused]] GpuIndex q, TQuery const& Q, GpuIndex leaf, GpuIndex i) const
    {
        auto inds = FromBuffers<4, 1>(tets, i);
        auto xe   = FromBuffers(verts, inds.Transpose());
        return DistanceQueries::PointTetrahedron(Q, xe.Col(0), xe.Col(1), xe.Col(2), xe.Col(3));
    }
};

struct FSetNearestNeighbour
{
    GpuIndex* NN;
    GpuScalar* dNN;
    PBAT_DEVICE void
    operator()(GpuIndex q, GpuIndex e, GpuScalar dmin, [[maybe_unused]] GpuIndex k) const
    {
        NN[q]  = e;
        dNN[q] = dmin;
    }
};

struct FDistanceUpperBound
{
    PBAT_DEVICE GpuScalar operator()(GpuIndex q) const
    {
        return std::numeric_limits<GpuScalar>::max();
    }
};

} // namespace Bvh
} // namespace test
} // namespace geometry
} // namespace impl
} // namespace gpu
} // namespace pbat

#include "pbat/gpu/impl/common/Eigen.cuh"

TEST_CASE("[gpu][impl][geometry] Bvh")
{
    using namespace pbat;
    // Cube mesh
    GpuMatrixX V(3, 8);
    GpuIndexMatrixX C(4, 5);
    // clang-format off
    V << 0.f, 1.f, 0.f, 1.f, 0.f, 1.f, 0.f, 1.f,
         0.f, 0.f, 1.f, 1.f, 0.f, 0.f, 1.f, 1.f,
         0.f, 0.f, 0.f, 0.f, 1.f, 1.f, 1.f, 1.f;
    C << 0, 3, 5, 6, 0,
         1, 2, 4, 7, 5,
         3, 0, 6, 5, 3,
         5, 6, 0, 3, 6;
    // clang-format on
    using gpu::impl::common::Buffer;
    using gpu::impl::common::ToBuffer;
    using gpu::impl::common::ToEigen;
    using gpu::impl::geometry::Aabb;
    using gpu::impl::geometry::Bvh;
    auto Vmin                = V.topRows<3>().rowwise().minCoeff().eval();
    auto Vmax                = V.topRows<3>().rowwise().maxCoeff().eval();
    using Overlap            = cuda::std::pair<GpuIndex, GpuIndex>;
    using Overlaps           = gpu::impl::common::SynchronizedList<Overlap>;
    using FOnOverlapDetected = gpu::impl::geometry::test::Bvh::FOnOverlapDetected;
    using namespace math::linalg;
    auto const fCheckCubeBvhTopology = [](Bvh const& bvh) {
        auto child = ToEigen(bvh.child).transpose().eval();
        CHECK_EQ(child.rows(), 4);
        CHECK_EQ(child.cols(), 2);
        CHECK_EQ(child(0, 0), 3);
        CHECK_EQ(child(0, 1), 8);
        CHECK_EQ(child(1, 0), 4);
        CHECK_EQ(child(1, 1), 5);
        CHECK_EQ(child(2, 0), 6);
        CHECK_EQ(child(2, 1), 7);
        CHECK_EQ(child(3, 0), 1);
        CHECK_EQ(child(3, 1), 2);
        auto parent = ToEigen(bvh.parent);
        CHECK_EQ(parent.rows(), 9);
        CHECK_EQ(parent.cols(), 1);
        CHECK_EQ(parent(0), GpuIndex{-1});
        CHECK_EQ(parent(1), 3);
        CHECK_EQ(parent(2), 3);
        CHECK_EQ(parent(3), 0);
        CHECK_EQ(parent(4), 1);
        CHECK_EQ(parent(5), 1);
        CHECK_EQ(parent(6), 2);
        CHECK_EQ(parent(7), 2);
        CHECK_EQ(parent(8), 0);
        auto rightmost       = ToEigen(bvh.rightmost).transpose().eval();
        auto const leafBegin = 4;
        CHECK_EQ(rightmost.rows(), 4);
        CHECK_EQ(rightmost.cols(), 2);
        CHECK_EQ(rightmost(0, 0), leafBegin + 3);
        CHECK_EQ(rightmost(0, 1), leafBegin + 4);
        CHECK_EQ(rightmost(1, 0), leafBegin + 0);
        CHECK_EQ(rightmost(1, 1), leafBegin + 1);
        CHECK_EQ(rightmost(2, 0), leafBegin + 2);
        CHECK_EQ(rightmost(2, 1), leafBegin + 3);
        CHECK_EQ(rightmost(3, 0), leafBegin + 1);
        CHECK_EQ(rightmost(3, 1), leafBegin + 3);
        auto visits = ToEigen(bvh.visits);
        CHECK_EQ(visits.rows(), 4);
        CHECK_EQ(visits.cols(), 1);
        bool const bTwoVisitsPerInternalNode = (visits.array() == 2).all();
        CHECK(bTwoVisitsPerInternalNode);
    };
    auto const fCheckInternalBoundingBoxComputation = [](Bvh const& bvh, Aabb<3> const& aabbs) {
        GpuIndexMatrixX child     = ToEigen(bvh.child);
        GpuMatrixX ib             = ToEigen(bvh.iaabbs.b);
        GpuMatrixX ie             = ToEigen(bvh.iaabbs.e);
        GpuMatrixX b              = ToEigen(aabbs.b);
        GpuMatrixX e              = ToEigen(aabbs.e);
        auto const nInternalNodes = bvh.iaabbs.Size();
        // For all internal nodes, check that their bounding box is the "union" of their children's
        for (auto i = 0; i < nInternalNodes; ++i)
        {
            auto lc           = child(0, i);
            auto rc           = child(1, i);
            auto lmin         = lc < nInternalNodes ? ib.col(lc).head<3>().eval() :
                                                      b.col(lc - nInternalNodes).head<3>().eval();
            auto lmax         = lc < nInternalNodes ? ie.col(lc).head<3>().eval() :
                                                      e.col(lc - nInternalNodes).head<3>().eval();
            auto rmin         = rc < nInternalNodes ? ib.col(rc).head<3>().eval() :
                                                      b.col(rc - nInternalNodes).head<3>().eval();
            auto rmax         = rc < nInternalNodes ? ie.col(rc).head<3>().eval() :
                                                      e.col(rc - nInternalNodes).head<3>().eval();
            auto iminExpected = lmin.cwiseMin(rmin).head<3>().eval();
            auto imaxExpected = lmax.cwiseMax(rmax).head<3>().eval();
            auto imin         = ib.col(i).head<3>().eval();
            auto imax         = ie.col(i).head<3>().eval();
            CHECK(imin.isApprox(iminExpected));
            CHECK(imax.isApprox(imaxExpected));
        }
    };
    SUBCASE("Connected non self-overlapping mesh")
    {
        // Arrange
        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Aabb<3> aabbs{VG, CG};
        Overlaps overlaps(1);
        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.DetectOverlaps(aabbs, FOnOverlapDetected{CG.Raw(), overlaps.Raw()});
        // Assert
        CHECK_EQ(overlaps.Size(), 0);
        fCheckCubeBvhTopology(bvh);
        fCheckInternalBoundingBoxComputation(bvh, aabbs);
    }
    SUBCASE("Disconnected mesh")
    {
        V = V(Eigen::placeholders::all, C.reshaped()).eval();
        C.resize(4, C.cols());
        C.reshaped().setLinSpaced(0, static_cast<GpuIndex>(V.cols() - 1));
        // Arrange
        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Aabb<3> aabbs{VG, CG};
        // Because we only support overlaps between i,j s.t. i<j to prevent duplicates, we use the
        // summation identity \sum_i=1^n i = n*(n+1)/2, and remove the n occurrences where i=j.
        auto const nSimplices        = aabbs.Size();
        auto const nExpectedOverlaps = (nSimplices * (nSimplices + 1) / 2) - nSimplices;
        Overlaps overlaps(2 * nExpectedOverlaps);
        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.DetectOverlaps(aabbs, FOnOverlapDetected{CG.Raw(), overlaps.Raw()});
        // Assert
        CHECK_EQ(overlaps.Size(), nExpectedOverlaps);
        fCheckCubeBvhTopology(bvh);
        fCheckInternalBoundingBoxComputation(bvh, aabbs);
    }
    SUBCASE("Non-overlapping line segment collection")
    {
        // Arrange
        // Bunch of disconnected line segments that do not overlap.
        V.setZero(3, 10);
        for (auto d = 0; d < 3; ++d)
        {
            V.row(d).setLinSpaced(GpuScalar(0), GpuScalar(9));
            V.row(d).reshaped(2, 5).row(1).array() -= GpuScalar(0.1);
        }
        Vmin = V.rowwise().minCoeff().eval();
        Vmax = V.rowwise().maxCoeff().eval();
        C.resize(4, 5);
        C.topRows(2).reshaped().setLinSpaced(0, static_cast<GpuIndex>(V.cols() - 1));
        C.bottomRows(2).reshaped().setLinSpaced(0, static_cast<GpuIndex>(V.cols() - 1));
        // Swap some columns of C to make sure that the order of simplices does not matter.
        C.col(0).swap(C.col(1));
        C.col(2).swap(C.col(3));
        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Aabb<3> aabbs{VG, CG};
        GpuIndex const nExpectedOverlaps{0};
        Overlaps overlaps(2 * nExpectedOverlaps);

        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.DetectOverlaps(aabbs, FOnOverlapDetected{CG.Raw(), overlaps.Raw()});

        // Assert
        CHECK_EQ(overlaps.Size(), nExpectedOverlaps);
        fCheckInternalBoundingBoxComputation(bvh, aabbs);
    }
    SUBCASE("Nearest neighbour search")
    {
        // Arrange
        GpuMatrixX QP(3, C.cols());
        QP.col(0) << GpuScalar(1.), GpuScalar(0.), GpuScalar(0.);
        QP.col(1) << GpuScalar(0.), GpuScalar(1.), GpuScalar(0.);
        QP.col(2) << GpuScalar(0.), GpuScalar(0.), GpuScalar(1.);
        QP.col(3) << GpuScalar(1.), GpuScalar(1.), GpuScalar(1.);
        QP.col(4) << GpuScalar(0.5), GpuScalar(0.5), GpuScalar(0.5);

        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Buffer<GpuScalar, 3> QPG(QP.cols());
        ToBuffer(QP, QPG);
        Buffer<GpuIndex> NNG(C.cols());
        NNG.SetConstant(GpuIndex(-1));
        Buffer<GpuScalar> dNNG(C.cols());
        dNNG.SetConstant(std::numeric_limits<GpuScalar>::max());
        Aabb<3> aabbs{VG, CG};

        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.NearestNeighbours(
            aabbs,
            static_cast<GpuIndex>(QP.cols()),
            gpu::impl::geometry::test::Bvh::FGetQueryObject{QPG.Raw()},
            gpu::impl::geometry::test::Bvh::FDistancePointAabb{},
            gpu::impl::geometry::test::Bvh::FDistancePointTetrahedron{VG.Raw(), CG.Raw()},
            gpu::impl::geometry::test::Bvh::FDistanceUpperBound{},
            gpu::impl::geometry::test::Bvh::FSetNearestNeighbour{NNG.Raw(), dNNG.Raw()});

        // Assert
        GpuIndexVectorX NN = ToEigen(NNG);
        GpuVectorX dNN     = ToEigen(dNNG);
        for (auto c = 0; c < C.cols(); ++c)
        {
            CHECK_EQ(NN(c), c);
            CHECK_EQ(dNN(c), GpuScalar(0));
        }
    }
}
