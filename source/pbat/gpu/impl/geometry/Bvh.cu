#include "hip/hip_runtime.h"
// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "Bvh.cuh"
#include "pbat/HostDevice.h"

#include <cuda/atomic>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <type_traits>

namespace pbat {
namespace gpu {
namespace impl {
namespace geometry {
namespace kernels {

namespace mini = pbat::math::linalg::mini;

struct FGenerateHierarchy
{
    using MortonCodeType = typename Bvh::MortonCodeType;

    struct Range
    {
        GpuIndex i, j, l;
        int d;
    };

    PBAT_DEVICE int Delta(GpuIndex i, GpuIndex j) const
    {
        if (j < 0 or j >= n)
            return -1;
        if (morton[i] == morton[j])
            return sizeof(MortonCodeType) * 8 /* #bits/byte */ + __clz(i ^ j);
        return __clz(morton[i] ^ morton[j]);
    }

    PBAT_DEVICE Range DetermineRange(GpuIndex i) const
    {
        // Compute range direction
        bool const dsign = (Delta(i, i + 1) - Delta(i, i - 1)) > 0;
        int const d      = 2 * dsign - 1;
        // Lower bound on length of internal node i's common prefix
        int const dmin = Delta(i, i - d);
        // Compute conservative upper bound on the range's size
        GpuIndex lmax{2};
        while (Delta(i, i + lmax * d) > dmin)
            lmax <<= 1;
        // Binary search in the "inflated" range for the actual end (or start) of internal node i's
        // range, considering that i is its start (or end).
        GpuIndex l{0};
        do
        {
            lmax >>= 1;
            if (Delta(i, i + (l + lmax) * d) > dmin)
                l += lmax;
        } while (lmax > 1);
        GpuIndex j = i + l * d;
        return Range{i, j, l, d};
    }

    PBAT_DEVICE GpuIndex FindSplit(Range R) const
    {
        // Calculate the number of highest bits that are the same
        // for all objects.
        int const dnode = Delta(R.i, R.j);

        // Use binary search to find where the next bit differs.
        // Specifically, we are looking for the highest object that
        // shares more than dnode bits with the first one.
        GpuIndex s{0};
        do
        {
            R.l = (R.l + 1) >> 1;
            if (Delta(R.i, R.i + (s + R.l) * R.d) > dnode)
                s += R.l;
        } while (R.l > 1);
        GpuIndex gamma = R.i + s * R.d + min(R.d, 0);
        return gamma;
    }

    PBAT_DEVICE void operator()(auto in)
    {
        // Find out which range of objects the node corresponds to.
        Range R = DetermineRange(in);
        // Determine where to split the range.
        GpuIndex gamma = FindSplit(R);
        // Select left+right child
        GpuIndex i  = min(R.i, R.j);
        GpuIndex j  = max(R.i, R.j);
        GpuIndex lc = (i == gamma) ? leafBegin + gamma : gamma;
        GpuIndex rc = (j == gamma + 1) ? leafBegin + gamma + 1 : gamma + 1;
        // Record parent-child relationships
        child[0][in] = lc;
        child[1][in] = rc;
        parent[lc]   = in;
        parent[rc]   = in;
        // Record subtree relationships
        rightmost[0][in] = leafBegin + gamma;
        rightmost[1][in] = leafBegin + j;
    }

    MortonCodeType const* morton;
    std::array<GpuIndex*, 2> child;
    GpuIndex* parent;
    std::array<GpuIndex*, 2> rightmost;
    GpuIndex leafBegin;
    GpuIndex n;
};

} // namespace kernels

Bvh::Bvh(GpuIndex nBoxes)
    : inds(nBoxes),
      morton(nBoxes),
      child(nBoxes - 1),
      parent(2 * nBoxes - 1),
      rightmost(nBoxes - 1),
      iaabbs(nBoxes - 1),
      visits(nBoxes - 1)
{
    parent.SetConstant(GpuIndex(-1));
}

void Bvh::Build(Aabb<kDims>& aabbs, Morton::Bound const& WL, Morton::Bound const& WU)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(ctx, "pbat.gpu.impl.geometry.Bvh.Build");
    using namespace pbat::math::linalg;
    GpuIndex const n                = aabbs.Size();
    GpuIndex const leafBegin        = n - 1;
    common::Buffer<GpuScalar, 3>& b = aabbs.b;
    common::Buffer<GpuScalar, 3>& e = aabbs.e;

    // 1. Reset intermediate data
    visits.SetConstant(GpuIndex(0));

    // 2. Compute Morton codes for each leaf node
    Morton::Encode(aabbs, WL, WU, morton);

    // 3. Sort leaves based on Morton codes
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(sortCtx, "pbat.gpu.impl.geometry.Bvh.Build.Sort");
    thrust::sequence(thrust::device, inds.Data(), inds.Data() + n);
    auto zip = thrust::make_zip_iterator(
        b[0].begin(),
        b[1].begin(),
        b[2].begin(),
        e[0].begin(),
        e[1].begin(),
        e[2].begin(),
        inds.Data());
    // Using a stable sort preserves the initial ordering of simplex indices 0...n-1, resulting in
    // simplices sorted by Morton codes first, and then by simplex index.
    thrust::stable_sort_by_key(thrust::device, morton.Data(), morton.Data() + n, zip);
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(sortCtx);

    // 4. Construct hierarchy
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(
        hierarchyCtx,
        "pbat.gpu.impl.geometry.Bvh.Build.Hierarchy");
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(n - 1),
        kernels::FGenerateHierarchy{
            morton.Raw(),
            child.Raw(),
            parent.Raw(),
            rightmost.Raw(),
            leafBegin,
            n});
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(hierarchyCtx);

    // 5. Construct internal node bounding boxes
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(
        iaabbCtx,
        "pbat.gpu.impl.geometry.Bvh.Build.InternalAabbs");
    auto& ib = iaabbs.b;
    auto& ie = iaabbs.e;
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(n - 1),
        thrust::make_counting_iterator(2 * n - 1),
        [leafBegin,
         parent = parent.Raw(),
         child  = child.Raw(),
         b      = b.Raw(),
         e      = e.Raw(),
         ib     = ib.Raw(),
         ie     = ie.Raw(),
         visits = visits.Raw()] PBAT_DEVICE(auto leaf) {
            auto p = parent[leaf];
            auto k = 0;
            for (; (k < 64) and (p >= 0); ++k)
            {
                cuda::atomic_ref<GpuIndex, cuda::thread_scope_device> ap{visits[p]};
                // The first thread that gets access to the internal node p will terminate,
                // while the second thread visiting p will be allowed to continue execution.
                // This ensures that there is no race condition where a thread can access an
                // internal node too early, i.e. before both children of the internal node
                // have finished computing their bounding boxes.
                if (ap++ == 0)
                    break;

                GpuIndex lc             = child[0][p];
                GpuIndex rc             = child[1][p];
                bool const bIsLeftLeaf  = lc >= leafBegin;
                bool const bIsRightLeaf = rc >= leafBegin;
                lc -= bIsLeftLeaf * leafBegin;
                rc -= bIsRightLeaf * leafBegin;
                auto bl = bIsLeftLeaf ? b : ib;
                auto el = bIsLeftLeaf ? e : ie;
                auto br = bIsRightLeaf ? b : ib;
                auto er = bIsRightLeaf ? e : ie;
                for (auto d = 0; d < 3; ++d)
                {
                    ib[d][p] = min(bl[d][lc], br[d][rc]);
                    ie[d][p] = max(el[d][lc], er[d][rc]);
                }
                // Move up the binary tree
                p = parent[p];
            }
            assert(k < 64);
        });
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(iaabbCtx);
    
    PBAT_PROFILE_CUDA_HOST_SCOPE_END(ctx);
}

} // namespace geometry
} // namespace impl
} // namespace gpu
} // namespace pbat

#include "pbat/common/ConstexprFor.h"
#include "pbat/common/Eigen.h"
#include "pbat/gpu/impl/common/SynchronizedList.cuh"

#include <algorithm>
#include <cuda/std/utility>
#include <doctest/doctest.h>
#include <unordered_set>

#pragma nv_diag_suppress 177

namespace pbat {
namespace gpu {
namespace impl {
namespace geometry {
namespace test {
namespace Bvh {

struct FOnOverlapDetected
{
    using Overlap = cuda::std::pair<GpuIndex, GpuIndex>;
    std::array<GpuIndex*, 4> T;
    common::DeviceSynchronizedList<Overlap> o;
    PBAT_DEVICE void operator()(GpuIndex si, GpuIndex sj)
    {
        using namespace pbat::math::linalg::mini;
        auto ti = FromBuffers<4, 1>(T, si);
        auto tj = FromBuffers<4, 1>(T, sj);
        bool bConnected{false};
        pbat::common::ForRange<0, 4>([&]<auto d>() { bConnected |= Any(tj == ti[d]); });
        if (not bConnected)
        {
            o.Append(Overlap{si, sj});
        }
    };
};

} // namespace Bvh
} // namespace test
} // namespace geometry
} // namespace impl
} // namespace gpu
} // namespace pbat

#include "pbat/gpu/impl/common/Eigen.cuh"

TEST_CASE("[gpu][impl][geometry] Bvh")
{
    using namespace pbat;
    // Cube mesh
    GpuMatrixX V(3, 8);
    GpuIndexMatrixX C(4, 5);
    // clang-format off
    V << 0.f, 1.f, 0.f, 1.f, 0.f, 1.f, 0.f, 1.f,
         0.f, 0.f, 1.f, 1.f, 0.f, 0.f, 1.f, 1.f,
         0.f, 0.f, 0.f, 0.f, 1.f, 1.f, 1.f, 1.f;
    C << 0, 3, 5, 6, 0,
         1, 2, 4, 7, 5,
         3, 0, 6, 5, 3,
         5, 6, 0, 3, 6;
    // clang-format on
    using gpu::impl::common::Buffer;
    using gpu::impl::common::ToBuffer;
    using gpu::impl::common::ToEigen;
    using gpu::impl::geometry::Aabb;
    using gpu::impl::geometry::Bvh;
    auto Vmin                = V.topRows<3>().rowwise().minCoeff().eval();
    auto Vmax                = V.topRows<3>().rowwise().maxCoeff().eval();
    using Overlap            = cuda::std::pair<GpuIndex, GpuIndex>;
    using Overlaps           = gpu::impl::common::SynchronizedList<Overlap>;
    using FOnOverlapDetected = gpu::impl::geometry::test::Bvh::FOnOverlapDetected;
    using namespace math::linalg;
    auto const fCheckCubeBvhTopology = [](Bvh const& bvh) {
        auto child = ToEigen(bvh.child).transpose().eval();
        CHECK_EQ(child.rows(), 4);
        CHECK_EQ(child.cols(), 2);
        CHECK_EQ(child(0, 0), 3);
        CHECK_EQ(child(0, 1), 8);
        CHECK_EQ(child(1, 0), 4);
        CHECK_EQ(child(1, 1), 5);
        CHECK_EQ(child(2, 0), 6);
        CHECK_EQ(child(2, 1), 7);
        CHECK_EQ(child(3, 0), 1);
        CHECK_EQ(child(3, 1), 2);
        auto parent = ToEigen(bvh.parent);
        CHECK_EQ(parent.rows(), 9);
        CHECK_EQ(parent.cols(), 1);
        CHECK_EQ(parent(0), GpuIndex{-1});
        CHECK_EQ(parent(1), 3);
        CHECK_EQ(parent(2), 3);
        CHECK_EQ(parent(3), 0);
        CHECK_EQ(parent(4), 1);
        CHECK_EQ(parent(5), 1);
        CHECK_EQ(parent(6), 2);
        CHECK_EQ(parent(7), 2);
        CHECK_EQ(parent(8), 0);
        auto rightmost       = ToEigen(bvh.rightmost).transpose().eval();
        auto const leafBegin = 4;
        CHECK_EQ(rightmost.rows(), 4);
        CHECK_EQ(rightmost.cols(), 2);
        CHECK_EQ(rightmost(0, 0), leafBegin + 3);
        CHECK_EQ(rightmost(0, 1), leafBegin + 4);
        CHECK_EQ(rightmost(1, 0), leafBegin + 0);
        CHECK_EQ(rightmost(1, 1), leafBegin + 1);
        CHECK_EQ(rightmost(2, 0), leafBegin + 2);
        CHECK_EQ(rightmost(2, 1), leafBegin + 3);
        CHECK_EQ(rightmost(3, 0), leafBegin + 1);
        CHECK_EQ(rightmost(3, 1), leafBegin + 3);
        auto visits = ToEigen(bvh.visits);
        CHECK_EQ(visits.rows(), 4);
        CHECK_EQ(visits.cols(), 1);
        bool const bTwoVisitsPerInternalNode = (visits.array() == 2).all();
        CHECK(bTwoVisitsPerInternalNode);
    };
    auto const fCheckInternalBoundingBoxComputation = [](Bvh const& bvh, Aabb<3> const& aabbs) {
        GpuIndexMatrixX child     = ToEigen(bvh.child);
        GpuMatrixX ib             = ToEigen(bvh.iaabbs.b);
        GpuMatrixX ie             = ToEigen(bvh.iaabbs.e);
        GpuMatrixX b              = ToEigen(aabbs.b);
        GpuMatrixX e              = ToEigen(aabbs.e);
        auto const nInternalNodes = bvh.iaabbs.Size();
        // For all internal nodes, check that their bounding box is the "union" of their children's
        for (auto i = 0; i < nInternalNodes; ++i)
        {
            auto lc           = child(0, i);
            auto rc           = child(1, i);
            auto lmin         = lc < nInternalNodes ? ib.col(lc).head<3>().eval() :
                                                      b.col(lc - nInternalNodes).head<3>().eval();
            auto lmax         = lc < nInternalNodes ? ie.col(lc).head<3>().eval() :
                                                      e.col(lc - nInternalNodes).head<3>().eval();
            auto rmin         = rc < nInternalNodes ? ib.col(rc).head<3>().eval() :
                                                      b.col(rc - nInternalNodes).head<3>().eval();
            auto rmax         = rc < nInternalNodes ? ie.col(rc).head<3>().eval() :
                                                      e.col(rc - nInternalNodes).head<3>().eval();
            auto iminExpected = lmin.cwiseMin(rmin).head<3>().eval();
            auto imaxExpected = lmax.cwiseMax(rmax).head<3>().eval();
            auto imin         = ib.col(i).head<3>().eval();
            auto imax         = ie.col(i).head<3>().eval();
            CHECK(imin.isApprox(iminExpected));
            CHECK(imax.isApprox(imaxExpected));
        }
    };
    SUBCASE("Connected non self-overlapping mesh")
    {
        // Arrange
        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Aabb<3> aabbs{VG, CG};
        Overlaps overlaps(1);
        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.DetectOverlaps(aabbs, FOnOverlapDetected{CG.Raw(), overlaps.Raw()});
        // Assert
        CHECK_EQ(overlaps.Size(), 0);
        fCheckCubeBvhTopology(bvh);
        fCheckInternalBoundingBoxComputation(bvh, aabbs);
    }
    SUBCASE("Disconnected mesh")
    {
        V = V(Eigen::placeholders::all, C.reshaped()).eval();
        C.resize(4, C.cols());
        C.reshaped().setLinSpaced(0, static_cast<GpuIndex>(V.cols() - 1));
        // Arrange
        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Aabb<3> aabbs{VG, CG};
        // Because we only support overlaps between i,j s.t. i<j to prevent duplicates, we use the
        // summation identity \sum_i=1^n i = n*(n+1)/2, and remove the n occurrences where i=j.
        auto const nSimplices        = aabbs.Size();
        auto const nExpectedOverlaps = (nSimplices * (nSimplices + 1) / 2) - nSimplices;
        Overlaps overlaps(2 * nExpectedOverlaps);
        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.DetectOverlaps(aabbs, FOnOverlapDetected{CG.Raw(), overlaps.Raw()});
        // Assert
        CHECK_EQ(overlaps.Size(), nExpectedOverlaps);
        fCheckCubeBvhTopology(bvh);
        fCheckInternalBoundingBoxComputation(bvh, aabbs);
    }
    SUBCASE("Non-overlapping line segment collection")
    {
        // Arrange
        // Bunch of disconnected line segments that do not overlap.
        V.setZero(3, 10);
        for (auto d = 0; d < 3; ++d)
        {
            V.row(d).setLinSpaced(GpuScalar(0), GpuScalar(9));
            V.row(d).reshaped(2, 5).row(1).array() -= GpuScalar(0.1);
        }
        Vmin = V.rowwise().minCoeff().eval();
        Vmax = V.rowwise().maxCoeff().eval();
        C.resize(4, 5);
        C.topRows(2).reshaped().setLinSpaced(0, static_cast<GpuIndex>(V.cols() - 1));
        C.bottomRows(2).reshaped().setLinSpaced(0, static_cast<GpuIndex>(V.cols() - 1));
        // Swap some columns of C to make sure that the order of simplices does not matter.
        C.col(0).swap(C.col(1));
        C.col(2).swap(C.col(3));
        Buffer<GpuScalar, 3> VG(V.cols());
        ToBuffer(V, VG);
        Buffer<GpuIndex, 4> CG(C.cols());
        ToBuffer(C, CG);
        Aabb<3> aabbs{VG, CG};
        GpuIndex const nExpectedOverlaps{0};
        Overlaps overlaps(2 * nExpectedOverlaps);

        // Act
        Bvh bvh(aabbs.Size());
        bvh.Build(aabbs, mini::FromEigen(Vmin), mini::FromEigen(Vmax));
        bvh.DetectOverlaps(aabbs, FOnOverlapDetected{CG.Raw(), overlaps.Raw()});

        // Assert
        CHECK_EQ(overlaps.Size(), nExpectedOverlaps);
        fCheckInternalBoundingBoxComputation(bvh, aabbs);
    }
}
