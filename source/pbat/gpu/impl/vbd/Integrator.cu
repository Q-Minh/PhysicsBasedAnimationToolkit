#include "hip/hip_runtime.h"
// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "Integrator.cuh"
#include "pbat/common/ConstexprFor.h"
#include "pbat/gpu/impl/common/Cuda.cuh"
#include "pbat/gpu/impl/common/Eigen.cuh"
#include "pbat/math/linalg/mini/Mini.h"
#include "pbat/profiling/Profiling.h"
#include "pbat/sim/vbd/Kernels.h"

#include <cuda/api.hpp>
// #include <thrust/async/copy.h>
#include <thrust/async/for_each.h>
#include <thrust/execution_policy.h>

namespace pbat {
namespace gpu {
namespace impl {
namespace vbd {

Integrator::Integrator(Data const& data)
    : x(data.x.cols()),
      T(data.E.cols()),
      mWorldMin(),
      mWorldMax(),
      cd(data.B.cast<GpuIndex>(), data.V.cast<GpuIndex>(), data.F.cast<GpuIndex>()),
      fc(data.x.cols() * kernels::BackwardEulerMinimization::kMaxCollidingTrianglesPerVertex),
      XVA(data.XVA.size()),
      FA(data.FA.size()),
      mActiveSetUpdateFrequency(static_cast<GpuIndex>(data.mActiveSetUpdateFrequency)),
      mPositionsAtT(data.xt.cols()),
      mInertialTargetPositions(data.xtilde.cols()),
      mChebyshevPositionsM2(data.xchebm2.cols()),
      mChebyshevPositionsM1(data.xchebm1.cols()),
      xb(data.x.cols()),
      mVelocitiesAtT(data.vt.cols()),
      mVelocities(data.v.cols()),
      mExternalAcceleration(data.aext.cols()),
      mMass(data.x.cols()),
      mQuadratureWeights(data.wg.size()),
      mShapeFunctionGradients(data.GP.size()),
      mLameCoefficients(data.lame.size()),
      mDetHZero(static_cast<GpuScalar>(data.detHZero)),
      mVertexTetrahedronPrefix(data.GVGp.size()),
      mVertexTetrahedronNeighbours(data.GVGe.size()),
      mVertexTetrahedronLocalVertexIndices(data.GVGilocal.size()),
      mRayleighDamping(static_cast<GpuScalar>(data.kD)),
      mCollisionPenalty(static_cast<GpuScalar>(data.muC)),
      mFrictionCoefficient(static_cast<GpuScalar>(data.muF)),
      mSmoothFrictionRelativeVelocityThreshold(static_cast<GpuScalar>(data.epsv)),
      mPptr(data.Pptr.cast<GpuIndex>()),
      mPadj(data.Padj.size()),
      mInitializationStrategy(data.strategy),
      mGpuThreadBlockSize(64),
      mStream(common::Device(common::EDeviceSelectionPreference::HighestComputeCapability)
                  .create_stream(/*synchronize_with_default_stream=*/false))
{
    common::ToBuffer(data.x, x);
    mPositionsAtT = x;
    common::ToBuffer(data.E, T);

    fc.SetConstant(GpuIndex(-1));
    common::ToBuffer(data.XVA, XVA);
    common::ToBuffer(data.FA, FA);

    common::ToBuffer(data.v, mVelocities);
    common::ToBuffer(data.aext, mExternalAcceleration);
    common::ToBuffer(data.m, mMass);

    common::ToBuffer(data.wg, mQuadratureWeights);
    common::ToBuffer(data.GP, mShapeFunctionGradients);
    common::ToBuffer(data.lame, mLameCoefficients);

    common::ToBuffer(data.GVGp, mVertexTetrahedronPrefix);
    mVertexTetrahedronNeighbours.Resize(data.GVGe.size());
    mVertexTetrahedronLocalVertexIndices.Resize(data.GVGilocal.size());
    common::ToBuffer(data.GVGe, mVertexTetrahedronNeighbours);
    common::ToBuffer(data.GVGilocal, mVertexTetrahedronLocalVertexIndices);

    common::ToBuffer(data.Padj.cast<GpuIndex>().eval(), mPadj);
}

void Integrator::Step(GpuScalar dt, GpuIndex iterations, GpuIndex substeps, GpuScalar rho)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(ctx, "pbat.gpu.impl.vbd.Integrator.Step");

    GpuScalar sdt  = dt / static_cast<GpuScalar>(substeps);
    GpuScalar sdt2 = sdt * sdt;

    InitializeActiveSet(dt);
    auto bdf = BdfDeviceParameters(sdt, sdt2);
    for (auto s = 0; s < substeps; ++s)
    {
        ComputeInertialTargets(sdt, sdt2);
        InitializeBcdSolution(sdt, sdt2);
        if (s % mActiveSetUpdateFrequency == 0)
            UpdateActiveSet();
        SolveBdfWithVbd(bdf, iterations, rho);
        UpdateBdfState(sdt);
    }
    cd.FinalizeActiveSet(x);

    PBAT_PROFILE_CUDA_HOST_SCOPE_END(ctx);
}

void Integrator::UpdateBdfState(GpuScalar sdt)
{
    GpuIndex const nVertices = static_cast<GpuIndex>(x.Size());
    mVelocitiesAtT           = mVelocities;
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator<GpuIndex>(0),
        thrust::make_counting_iterator<GpuIndex>(nVertices),
        [xt = mPositionsAtT.Raw(), x = x.Raw(), v = mVelocities.Raw(), dt = sdt] PBAT_DEVICE(
            auto i) {
            using pbat::sim::vbd::kernels::IntegrateVelocity;
            using pbat::math::linalg::mini::FromBuffers;
            using pbat::math::linalg::mini::ToBuffers;
            auto vtp1 = IntegrateVelocity(FromBuffers<3, 1>(xt, i), FromBuffers<3, 1>(x, i), dt);
            ToBuffers(vtp1, v, i);
        });
    mPositionsAtT = x;
}

kernels::BackwardEulerMinimization Integrator::BdfDeviceParameters(GpuScalar dt, GpuScalar dt2)
{
    kernels::BackwardEulerMinimization bdf{};
    bdf.dt        = dt;
    bdf.dt2       = dt2;
    bdf.m         = mMass.Raw();
    bdf.xtilde    = mInertialTargetPositions.Raw();
    bdf.xt        = mPositionsAtT.Raw();
    bdf.x         = x.Raw();
    bdf.xb        = xb.Raw();
    bdf.T         = T.Raw();
    bdf.wg        = mQuadratureWeights.Raw();
    bdf.GP        = mShapeFunctionGradients.Raw();
    bdf.lame      = mLameCoefficients.Raw();
    bdf.detHZero  = mDetHZero;
    bdf.GVTp      = mVertexTetrahedronPrefix.Raw();
    bdf.GVTn      = mVertexTetrahedronNeighbours.Raw();
    bdf.GVTilocal = mVertexTetrahedronLocalVertexIndices.Raw();
    bdf.kD        = mRayleighDamping;
    bdf.muC       = mCollisionPenalty;
    bdf.muF       = mFrictionCoefficient;
    bdf.epsv      = mSmoothFrictionRelativeVelocityThreshold;
    bdf.fc        = fc.Raw();
    bdf.F         = cd.F.Raw();
    bdf.XVA       = XVA.Raw();
    bdf.FA        = FA.Raw();
    return bdf;
}

void Integrator::SetPositions(Eigen::Ref<GpuMatrixX const> const& Xin)
{
    common::ToBuffer(Xin, x);
    mPositionsAtT = x;
}

void Integrator::SetVelocities(Eigen::Ref<GpuMatrixX const> const& v)
{
    common::ToBuffer(v, mVelocities);
    mVelocitiesAtT = mVelocities;
}

void Integrator::SetExternalAcceleration(Eigen::Ref<GpuMatrixX const> const& aext)
{
    common::ToBuffer(aext, mExternalAcceleration);
}

void Integrator::SetMass(Eigen::Ref<GpuVectorX const> const& m)
{
    common::ToBuffer(m, mMass);
}

void Integrator::SetQuadratureWeights(Eigen::Ref<GpuVectorX const> const& wg)
{
    common::ToBuffer(wg, mQuadratureWeights);
}

void Integrator::SetShapeFunctionGradients(Eigen::Ref<GpuMatrixX const> const& GP)
{
    common::ToBuffer(GP, mShapeFunctionGradients);
}

void Integrator::SetLameCoefficients(Eigen::Ref<GpuMatrixX const> const& l)
{
    common::ToBuffer(l, mLameCoefficients);
}

void Integrator::SetNumericalZeroForHessianDeterminant(GpuScalar zero)
{
    mDetHZero = zero;
}

void Integrator::SetVertexTetrahedronAdjacencyList(
    Eigen::Ref<GpuIndexVectorX const> const& GVTp,
    Eigen::Ref<GpuIndexVectorX const> const& GVTn,
    Eigen::Ref<GpuIndexVectorX const> const& GVTilocal)
{
    if (GVTn.size() != GVTilocal.size())
    {
        std::ostringstream ss{};
        ss << "Expected vertex-tetrahedron adjacency graph's neighbour array and data (ilocal) "
              "array to have the same size, but got neighbours="
           << GVTn.size() << ", ilocal=" << GVTilocal.size() << " \n";
        throw std::invalid_argument(ss.str());
    }

    common::ToBuffer(GVTp, mVertexTetrahedronPrefix);
    mVertexTetrahedronNeighbours.Resize(GVTn.size());
    mVertexTetrahedronLocalVertexIndices.Resize(GVTilocal.size());
    common::ToBuffer(GVTn, mVertexTetrahedronNeighbours);
    common::ToBuffer(GVTilocal, mVertexTetrahedronLocalVertexIndices);
}

void Integrator::SetRayleighDampingCoefficient(GpuScalar kD)
{
    mRayleighDamping = kD;
}

void Integrator::SetVertexPartitions(
    Eigen::Ref<GpuIndexVectorX const> const& Pptr,
    Eigen::Ref<GpuIndexVectorX const> const& Padj)
{
    mPptr = Pptr;
    common::ToBuffer(Padj, mPadj);
}

void Integrator::SetInitializationStrategy(EInitializationStrategy strategy)
{
    mInitializationStrategy = strategy;
}

void Integrator::SetSceneBoundingBox(
    Eigen::Vector<GpuScalar, 3> const& min,
    Eigen::Vector<GpuScalar, 3> const& max)
{
    mWorldMin = min;
    mWorldMax = max;
}

void Integrator::SetBlockSize(GpuIndex blockSize)
{
    mGpuThreadBlockSize = std::clamp(blockSize, GpuIndex{32}, GpuIndex{256});
}

common::Buffer<GpuScalar, 3> const& Integrator::GetVelocity() const
{
    return mVelocities;
}

common::Buffer<GpuScalar, 3> const& Integrator::GetExternalAcceleration() const
{
    return mExternalAcceleration;
}

common::Buffer<GpuScalar> const& Integrator::GetMass() const
{
    return mMass;
}

common::Buffer<GpuScalar> const& Integrator::GetShapeFunctionGradients() const
{
    return mShapeFunctionGradients;
}

common::Buffer<GpuScalar> const& Integrator::GetLameCoefficients() const
{
    return mLameCoefficients;
}

void Integrator::InitializeActiveSet(GpuScalar dt)
{
    GpuIndex const nVertices = static_cast<GpuIndex>(x.Size());
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator<GpuIndex>(0),
        thrust::make_counting_iterator<GpuIndex>(nVertices),
        [dt,
         dt2      = dt * dt,
         xt       = mPositionsAtT.Raw(),
         vt       = mVelocities.Raw(),
         aext     = mExternalAcceleration.Raw(),
         x        = x.Raw(),
         strategy = mInitializationStrategy] PBAT_DEVICE(auto i) {
            using namespace pbat::math::linalg::mini;
            auto xti   = FromBuffers<3, 1>(xt, i);
            auto vti   = FromBuffers<3, 1>(vt, i);
            auto aexti = FromBuffers<3, 1>(aext, i);
            auto xi    = xti + dt * vti + dt2 * aexti;
            ToBuffers(xi, x, i);
        });
    using pbat::math::linalg::mini::FromEigen;
    cd.InitializeActiveSet(mPositionsAtT, x, FromEigen(mWorldMin), FromEigen(mWorldMax));
}

void Integrator::ComputeInertialTargets(GpuScalar sdt, GpuScalar sdt2)
{
    GpuIndex const nVertices = static_cast<GpuIndex>(x.Size());
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator<GpuIndex>(0),
        thrust::make_counting_iterator<GpuIndex>(nVertices),
        [xt     = mPositionsAtT.Raw(),
         vt     = mVelocities.Raw(),
         aext   = mExternalAcceleration.Raw(),
         xtilde = mInertialTargetPositions.Raw(),
         dt     = sdt,
         dt2    = sdt2] PBAT_DEVICE(auto i) {
            using pbat::sim::vbd::kernels::InertialTarget;
            using pbat::math::linalg::mini::FromBuffers;
            using pbat::math::linalg::mini::ToBuffers;
            auto y = InertialTarget(
                FromBuffers<3, 1>(xt, i),
                FromBuffers<3, 1>(vt, i),
                FromBuffers<3, 1>(aext, i),
                dt,
                dt2);
            ToBuffers(y, xtilde, i);
        });
}

void Integrator::InitializeBcdSolution(GpuScalar sdt, GpuScalar sdt2)
{
    GpuIndex const nVertices = static_cast<GpuIndex>(x.Size());
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator<GpuIndex>(0),
        thrust::make_counting_iterator<GpuIndex>(nVertices),
        [xt       = mPositionsAtT.Raw(),
         vtm1     = mVelocitiesAtT.Raw(),
         vt       = mVelocities.Raw(),
         aext     = mExternalAcceleration.Raw(),
         x        = x.Raw(),
         dt       = sdt,
         dt2      = sdt2,
         strategy = mInitializationStrategy] PBAT_DEVICE(auto i) {
            using pbat::sim::vbd::kernels::InitialPositionsForSolve;
            using pbat::math::linalg::mini::FromBuffers;
            using pbat::math::linalg::mini::ToBuffers;
            auto x0 = InitialPositionsForSolve(
                FromBuffers<3, 1>(xt, i),
                FromBuffers<3, 1>(vtm1, i),
                FromBuffers<3, 1>(vt, i),
                FromBuffers<3, 1>(aext, i),
                dt,
                dt2,
                strategy);
            ToBuffers(x0, x, i);
        });
}

void Integrator::UpdateActiveSet()
{
    cd.UpdateActiveSet(x);
    static auto constexpr kMaxContacts =
        kernels::BackwardEulerMinimization::kMaxCollidingTrianglesPerVertex;
    static auto constexpr kMaxNeighbours = contact::VertexTriangleMixedCcdDcd::kMaxNeighbours;
    thrust::for_each(
        thrust::device,
        cd.av.Data(),
        cd.av.Data() + cd.nActive,
        [V = cd.V.Raw(), nn = cd.nn.Raw(), fc = fc.Raw()] PBAT_DEVICE(GpuIndex v) {
            using namespace pbat::math::linalg::mini;
            GpuIndex i                            = V[v];
            SVector<GpuIndex, kMaxNeighbours> nnv = FromFlatBuffer<kMaxNeighbours, 1>(nn, v);
            SVector<GpuIndex, kMaxContacts> f     = -Ones<GpuIndex, kMaxContacts, 1>();
            auto const top                        = min(kMaxContacts, kMaxNeighbours);
            for (auto c = 0; c < top; ++c)
                if (nnv(c) >= 0)
                    f(c) = nnv(c);
            ToFlatBuffer(f, fc, i);
        });
}

void Integrator::SolveBdfWithVbd(
    kernels::BackwardEulerMinimization& bdf,
    GpuIndex iterations,
    GpuScalar rho)
{
    bool bUseChebyshevAcceleration = rho > GpuScalar{0} and rho < GpuScalar{1};
    GpuScalar rho2                 = rho * rho;
    GpuScalar omega{};
    GpuIndex const nVertices = static_cast<GpuIndex>(x.Size());

    for (auto k = 0; k < iterations; ++k)
    {
        using pbat::sim::vbd::kernels::ChebyshevOmega;
        if (bUseChebyshevAcceleration)
            omega = ChebyshevOmega(k, rho2, omega);

        auto const nPartitions = mPptr.size() - 1;
        for (auto p = 0; p < nPartitions; ++p)
        {
            auto pBegin                     = mPptr[p];
            auto pEnd                       = mPptr[p + 1];
            bdf.partition                   = mPadj.Raw() + pBegin;
            auto const nVerticesInPartition = static_cast<cuda::grid::dimension_t>(pEnd - pBegin);
            pbat::common::ForValues<32, 64, 128, 256>([&]<auto kBlockThreads>() {
                if (mGpuThreadBlockSize > kBlockThreads / 2 and
                    mGpuThreadBlockSize <= kBlockThreads)
                {
                    auto const kDynamicSharedMemoryCapacity = static_cast<
                        cuda::memory::shared::size_t>(sizeof(
                        typename kernels::BackwardEulerMinimization::BlockStorage<kBlockThreads>));
                    auto bcdLaunchConfiguration =
                        cuda::launch_config_builder()
                            .block_size(kBlockThreads)
                            .dynamic_shared_memory_size(kDynamicSharedMemoryCapacity)
                            .grid_size(nVerticesInPartition)
                            .build();
                    cuda::device::current::get().launch(
                        kernels::MinimizeBackwardEuler<kBlockThreads>,
                        bcdLaunchConfiguration,
                        bdf);
                }
            });
            // Copy xb back to x
            thrust::for_each(
                thrust::device,
                bdf.partition,
                bdf.partition + nVerticesInPartition,
                [xb = xb.Raw(), x = x.Raw()] PBAT_DEVICE(GpuIndex i) {
                    using namespace pbat::math::linalg::mini;
                    ToBuffers(FromBuffers<3, 1>(xb, i), x, i);
                });
        }

        if (bUseChebyshevAcceleration)
        {
            thrust::for_each(
                thrust::device,
                thrust::make_counting_iterator<GpuIndex>(0),
                thrust::make_counting_iterator<GpuIndex>(nVertices),
                [k     = k,
                 omega = omega,
                 xkm2  = mChebyshevPositionsM2.Raw(),
                 xkm1  = mChebyshevPositionsM1.Raw(),
                 xk    = x.Raw()] PBAT_DEVICE(auto i) {
                    using pbat::sim::vbd::kernels::ChebyshevUpdate;
                    using pbat::math::linalg::mini::FromBuffers;
                    using pbat::math::linalg::mini::ToBuffers;
                    auto xkm2i = FromBuffers<3, 1>(xkm2, i);
                    auto xkm1i = FromBuffers<3, 1>(xkm1, i);
                    auto xki   = FromBuffers<3, 1>(xk, i);
                    ChebyshevUpdate(k, omega, xkm2i, xkm1i, xki);
                    ToBuffers(xkm2i, xkm2, i);
                    ToBuffers(xkm1i, xkm1, i);
                    ToBuffers(xki, xk, i);
                });
        }
    }
}

} // namespace vbd
} // namespace impl
} // namespace gpu
} // namespace pbat

#include "pbat/common/Eigen.h"

#include <Eigen/SparseCore>
#include <doctest/doctest.h>
#include <span>
#include <vector>

TEST_CASE("[gpu][impl][vbd] Integrator")
{
    using namespace pbat;
    using pbat::common::ToEigen;
    // Arrange
    // Cube mesh
    MatrixX P(3, 8);
    IndexMatrixX V(1, 8);
    IndexMatrixX T(4, 5);
    IndexMatrixX F(3, 12);
    // clang-format off
    P << 0.f, 1.f, 0.f, 1.f, 0.f, 1.f, 0.f, 1.f,
         0.f, 0.f, 1.f, 1.f, 0.f, 0.f, 1.f, 1.f,
         0.f, 0.f, 0.f, 0.f, 1.f, 1.f, 1.f, 1.f;
    T << 0, 3, 5, 6, 0,
         1, 2, 4, 7, 5,
         3, 0, 6, 5, 3,
         5, 6, 0, 3, 6;
    F << 0, 1, 1, 3, 3, 2, 2, 0, 0, 0, 4, 5,
         1, 5, 3, 7, 2, 6, 0, 4, 3, 2, 5, 7,
         4, 4, 5, 5, 7, 7, 6, 6, 1, 3, 6, 6;
    // clang-format on
    V.reshaped().setLinSpaced(0, static_cast<Index>(P.cols() - 1));
    // Problem parameters
    auto constexpr dt         = GpuScalar{1e-2};
    auto constexpr substeps   = 1;
    auto constexpr iterations = 10;
    auto const worldMin       = P.rowwise().minCoeff().cast<GpuScalar>().eval();
    auto const worldMax       = P.rowwise().maxCoeff().cast<GpuScalar>().eval();

    // Act
    using pbat::gpu::impl::vbd::Integrator;
    Integrator vbd{sim::vbd::Data()
                       .WithVolumeMesh(P, T)
                       .WithSurfaceMesh(V, F)
                       .WithBodies(IndexVectorX::Ones(P.cols()))
                       .Construct()};
    vbd.SetSceneBoundingBox(worldMin, worldMax);
    vbd.Step(dt, iterations, substeps);

    // Assert
    auto constexpr zero = GpuScalar{1e-4};
    GpuMatrixX dx =
        ToEigen(vbd.x.Get()).reshaped(P.cols(), P.rows()).transpose() - P.cast<GpuScalar>();
    bool const bVerticesFallUnderGravity = (dx.row(2).array() < GpuScalar{0}).all();
    CHECK(bVerticesFallUnderGravity);
    bool const bVerticesOnlyFall = (dx.topRows(2).array().abs() < zero).all();
    CHECK(bVerticesOnlyFall);
}
