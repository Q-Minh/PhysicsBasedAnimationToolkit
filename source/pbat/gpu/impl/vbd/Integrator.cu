#include "hip/hip_runtime.h"
// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "Integrator.cuh"
#include "Kernels.cuh"
#include "pbat/common/ConstexprFor.h"
#include "pbat/gpu/impl/common/Cuda.cuh"
#include "pbat/gpu/impl/common/Eigen.cuh"
#include "pbat/math/linalg/mini/Mini.h"
#include "pbat/profiling/Profiling.h"
#include "pbat/sim/vbd/Kernels.h"

#include <cuda/api.hpp>
// #include <thrust/async/copy.h>
#include <thrust/async/for_each.h>
#include <thrust/execution_policy.h>

namespace pbat {
namespace gpu {
namespace impl {
namespace vbd {

Integrator::Integrator(Data const& data)
    : X(data.x.cast<GpuScalar>()),
      V(data.V.cast<GpuIndex>().transpose()),
      F(data.F.cast<GpuIndex>()),
      T(data.E.cast<GpuIndex>()),
      mPositionsAtT(data.xt.cols()),
      mInertialTargetPositions(data.xtilde.cols()),
      mChebyshevPositionsM2(data.xchebm2.cols()),
      mChebyshevPositionsM1(data.xchebm1.cols()),
      mVelocitiesAtT(data.vt.cols()),
      mVelocities(data.v.cols()),
      mExternalAcceleration(data.aext.cols()),
      mMass(data.x.cols()),
      mQuadratureWeights(data.wg.size()),
      mShapeFunctionGradients(data.GP.size()),
      mLameCoefficients(data.lame.size()),
      mDetHZero(static_cast<GpuScalar>(data.detHZero)),
      mVertexTetrahedronPrefix(data.GVGp.size()),
      mVertexTetrahedronNeighbours(data.GVGe.size()),
      mVertexTetrahedronLocalVertexIndices(data.GVGilocal.size()),
      mRayleighDamping(static_cast<GpuScalar>(data.kD)),
      mCollisionPenalty(static_cast<GpuScalar>(data.kC)),
      mMaxCollidingTrianglesPerVertex(8),
      mCollidingTriangles(8 * data.x.cols()),
      mCollidingTriangleCount(data.x.cols()),
      mPptr(data.Pptr.cast<GpuIndex>()),
      mPadj(data.Padj.size()),
      mInitializationStrategy(data.strategy),
      mGpuThreadBlockSize(64),
      mStream(common::Device(common::EDeviceSelectionPreference::HighestComputeCapability)
                  .create_stream(/*synchronize_with_default_stream=*/false))
{
    common::ToBuffer(data.v, mVelocities);
    common::ToBuffer(data.aext, mExternalAcceleration);
    common::ToBuffer(data.m, mMass);

    common::ToBuffer(data.wg, mQuadratureWeights);
    common::ToBuffer(data.GP, mShapeFunctionGradients);
    common::ToBuffer(data.lame, mLameCoefficients);

    common::ToBuffer(data.GVGp, mVertexTetrahedronPrefix);
    mVertexTetrahedronNeighbours.Resize(data.GVGe.size());
    mVertexTetrahedronLocalVertexIndices.Resize(data.GVGilocal.size());
    common::ToBuffer(data.GVGe, mVertexTetrahedronNeighbours);
    common::ToBuffer(data.GVGilocal, mVertexTetrahedronLocalVertexIndices);

    common::ToBuffer(data.Padj.cast<GpuIndex>().eval(), mPadj);
}

void Integrator::Step(GpuScalar dt, GpuIndex iterations, GpuIndex substeps, GpuScalar rho)
{
    PBAT_PROFILE_NAMED_CUDA_HOST_SCOPE_START(ctx, "pbat.gpu.impl.vbd.Integrator.Step");

    GpuScalar sdt                        = dt / static_cast<GpuScalar>(substeps);
    GpuScalar sdt2                       = sdt * sdt;
    GpuIndex const nVertices             = static_cast<GpuIndex>(X.NumberOfPoints());
    bool const bUseChebyshevAcceleration = rho > GpuScalar{0} and rho < GpuScalar{1};

    kernels::BackwardEulerMinimization bdf{};
    bdf.dt                              = sdt;
    bdf.dt2                             = sdt2;
    bdf.m                               = mMass.Raw();
    bdf.xtilde                          = mInertialTargetPositions.Raw();
    bdf.xt                              = mPositionsAtT.Raw();
    bdf.x                               = X.x.Raw();
    bdf.T                               = T.inds.Raw();
    bdf.wg                              = mQuadratureWeights.Raw();
    bdf.GP                              = mShapeFunctionGradients.Raw();
    bdf.lame                            = mLameCoefficients.Raw();
    bdf.detHZero                        = mDetHZero;
    bdf.GVTp                            = mVertexTetrahedronPrefix.Raw();
    bdf.GVTn                            = mVertexTetrahedronNeighbours.Raw();
    bdf.GVTilocal                       = mVertexTetrahedronLocalVertexIndices.Raw();
    bdf.kD                              = mRayleighDamping;
    bdf.kC                              = mCollisionPenalty;
    bdf.nMaxCollidingTrianglesPerVertex = mMaxCollidingTrianglesPerVertex;
    bdf.FC                              = mCollidingTriangles.Raw();
    bdf.nCollidingTriangles             = mCollidingTriangleCount.Raw();
    bdf.F                               = F.inds.Raw();

    // NOTE:
    // For some reason, thrust::async::copy does not play well with cuda-api-wrapper streams. I am
    // guessing it has to do with synchronize_with_default_stream=false?
    mStream.device().make_current();
    for (auto s = 0; s < substeps; ++s)
    {
        using namespace pbat::math::linalg::mini;
        // Store previous positions
        for (auto d = 0; d < X.x.Dimensions(); ++d)
        {
            cuda::memory::async::copy(
                thrust::raw_pointer_cast(mPositionsAtT[d].data()),
                thrust::raw_pointer_cast(X.x[d].data()),
                X.x.Size() * sizeof(GpuScalar),
                mStream);
        }
        // Compute inertial target positions
        thrust::device_event e = thrust::async::for_each(
            // Share thrust's underlying CUDA stream with cuda-api-wrappers
            thrust::device.on(mStream.handle()),
            thrust::make_counting_iterator<GpuIndex>(0),
            thrust::make_counting_iterator<GpuIndex>(nVertices),
            [xt     = mPositionsAtT.Raw(),
             vt     = mVelocities.Raw(),
             aext   = mExternalAcceleration.Raw(),
             xtilde = mInertialTargetPositions.Raw(),
             dt     = sdt,
             dt2    = sdt2] PBAT_DEVICE(auto i) {
                using pbat::sim::vbd::kernels::InertialTarget;
                auto y = InertialTarget(
                    FromBuffers<3, 1>(xt, i),
                    FromBuffers<3, 1>(vt, i),
                    FromBuffers<3, 1>(aext, i),
                    dt,
                    dt2);
                ToBuffers(y, xtilde, i);
            });
        // Initialize block coordinate descent's, i.e. BCD's, solution
        e = thrust::async::for_each(
            thrust::device.on(mStream.handle()),
            thrust::make_counting_iterator<GpuIndex>(0),
            thrust::make_counting_iterator<GpuIndex>(nVertices),
            [xt       = mPositionsAtT.Raw(),
             vtm1     = mVelocitiesAtT.Raw(),
             vt       = mVelocities.Raw(),
             aext     = mExternalAcceleration.Raw(),
             x        = X.x.Raw(),
             dt       = sdt,
             dt2      = sdt2,
             strategy = mInitializationStrategy] PBAT_DEVICE(auto i) {
                using pbat::sim::vbd::kernels::InitialPositionsForSolve;
                auto x0 = InitialPositionsForSolve(
                    FromBuffers<3, 1>(xt, i),
                    FromBuffers<3, 1>(vtm1, i),
                    FromBuffers<3, 1>(vt, i),
                    FromBuffers<3, 1>(aext, i),
                    dt,
                    dt2,
                    strategy);
                ToBuffers(x0, x, i);
            });
        // Initialize Chebyshev semi-iterative method
        GpuScalar rho2 = rho * rho;
        GpuScalar omega{};
        // Minimize Backward Euler, i.e. BDF1, objective
        for (auto k = 0; k < iterations; ++k)
        {
            using pbat::sim::vbd::kernels::ChebyshevOmega;
            if (bUseChebyshevAcceleration)
                omega = ChebyshevOmega(k, rho2, omega);

            auto const nPartitions = mPptr.size() - 1;
            for (auto p = 0; p < nPartitions; ++p)
            {
                auto pBegin   = mPptr[p];
                auto pEnd     = mPptr[p + 1];
                bdf.partition = mPadj.Raw() + pBegin;
                auto const nVerticesInPartition =
                    static_cast<cuda::grid::dimension_t>(pEnd - pBegin);
                pbat::common::ForValues<32, 64, 128, 256>([&]<auto kBlockThreads>() {
                    if (mGpuThreadBlockSize > kBlockThreads / 2 and
                        mGpuThreadBlockSize <= kBlockThreads)
                    {
                        auto const kDynamicSharedMemoryCapacity =
                            static_cast<cuda::memory::shared::size_t>(
                                sizeof(typename kernels::BackwardEulerMinimization::BlockStorage<
                                       kBlockThreads>));
                        auto bcdLaunchConfiguration =
                            cuda::launch_config_builder()
                                .block_size(kBlockThreads)
                                .dynamic_shared_memory_size(kDynamicSharedMemoryCapacity)
                                .grid_size(nVerticesInPartition)
                                .build();
                        mStream.enqueue.kernel_launch(
                            kernels::MinimizeBackwardEuler<kBlockThreads>,
                            bcdLaunchConfiguration,
                            bdf);
                    }
                });
            }

            if (bUseChebyshevAcceleration)
            {
                e = thrust::async::for_each(
                    thrust::device.on(mStream.handle()),
                    thrust::make_counting_iterator<GpuIndex>(0),
                    thrust::make_counting_iterator<GpuIndex>(nVertices),
                    [k     = k,
                     omega = omega,
                     xkm2  = mChebyshevPositionsM2.Raw(),
                     xkm1  = mChebyshevPositionsM1.Raw(),
                     xk    = X.x.Raw()] PBAT_DEVICE(auto i) {
                        using pbat::sim::vbd::kernels::ChebyshevUpdate;
                        auto xkm2i = FromBuffers<3, 1>(xkm2, i);
                        auto xkm1i = FromBuffers<3, 1>(xkm1, i);
                        auto xki   = FromBuffers<3, 1>(xk, i);
                        ChebyshevUpdate(k, omega, xkm2i, xkm1i, xki);
                        ToBuffers(xkm2i, xkm2, i);
                        ToBuffers(xkm1i, xkm1, i);
                        ToBuffers(xki, xk, i);
                    });
            }
        }
        // Update velocities
        for (auto d = 0; d < mVelocities.Dimensions(); ++d)
        {
            cuda::memory::async::copy(
                thrust::raw_pointer_cast(mVelocitiesAtT[d].data()),
                thrust::raw_pointer_cast(mVelocities[d].data()),
                mVelocities.Size() * sizeof(GpuScalar),
                mStream);
        }
        e = thrust::async::for_each(
            thrust::device.on(mStream.handle()),
            thrust::make_counting_iterator<GpuIndex>(0),
            thrust::make_counting_iterator<GpuIndex>(nVertices),
            [xt = mPositionsAtT.Raw(), x = X.x.Raw(), v = mVelocities.Raw(), dt = sdt] PBAT_DEVICE(
                auto i) {
                using pbat::sim::vbd::kernels::IntegrateVelocity;
                auto vtp1 =
                    IntegrateVelocity(FromBuffers<3, 1>(xt, i), FromBuffers<3, 1>(x, i), dt);
                ToBuffers(vtp1, v, i);
            });
    }
    mStream.synchronize();

    PBAT_PROFILE_CUDA_HOST_SCOPE_END(ctx);
}

void Integrator::SetPositions(Eigen::Ref<GpuMatrixX const> const& Xin)
{
    common::ToBuffer(Xin, X.x);
}

void Integrator::SetVelocities(Eigen::Ref<GpuMatrixX const> const& v)
{
    common::ToBuffer(v, mVelocities);
}

void Integrator::SetExternalAcceleration(Eigen::Ref<GpuMatrixX const> const& aext)
{
    common::ToBuffer(aext, mExternalAcceleration);
}

void Integrator::SetMass(Eigen::Ref<GpuVectorX const> const& m)
{
    common::ToBuffer(m, mMass);
}

void Integrator::SetQuadratureWeights(Eigen::Ref<GpuVectorX const> const& wg)
{
    common::ToBuffer(wg, mQuadratureWeights);
}

void Integrator::SetShapeFunctionGradients(Eigen::Ref<GpuMatrixX const> const& GP)
{
    common::ToBuffer(GP, mShapeFunctionGradients);
}

void Integrator::SetLameCoefficients(Eigen::Ref<GpuMatrixX const> const& l)
{
    common::ToBuffer(l, mLameCoefficients);
}

void Integrator::SetNumericalZeroForHessianDeterminant(GpuScalar zero)
{
    mDetHZero = zero;
}

void Integrator::SetVertexTetrahedronAdjacencyList(
    Eigen::Ref<GpuIndexVectorX const> const& GVTp,
    Eigen::Ref<GpuIndexVectorX const> const& GVTn,
    Eigen::Ref<GpuIndexVectorX const> const& GVTilocal)
{
    if (GVTn.size() != GVTilocal.size())
    {
        std::ostringstream ss{};
        ss << "Expected vertex-tetrahedron adjacency graph's neighbour array and data (ilocal) "
              "array to have the same size, but got neighbours="
           << GVTn.size() << ", ilocal=" << GVTilocal.size() << " \n";
        throw std::invalid_argument(ss.str());
    }

    common::ToBuffer(GVTp, mVertexTetrahedronPrefix);
    mVertexTetrahedronNeighbours.Resize(GVTn.size());
    mVertexTetrahedronLocalVertexIndices.Resize(GVTilocal.size());
    common::ToBuffer(GVTn, mVertexTetrahedronNeighbours);
    common::ToBuffer(GVTilocal, mVertexTetrahedronLocalVertexIndices);
}

void Integrator::SetRayleighDampingCoefficient(GpuScalar kD)
{
    mRayleighDamping = kD;
}

void Integrator::SetVertexPartitions(
    Eigen::Ref<GpuIndexVectorX const> const& Pptr,
    Eigen::Ref<GpuIndexVectorX const> const& Padj)
{
    mPptr = Pptr;
    common::ToBuffer(Padj, mPadj);
}

void Integrator::SetInitializationStrategy(EInitializationStrategy strategy)
{
    mInitializationStrategy = strategy;
}

void Integrator::SetBlockSize(GpuIndex blockSize)
{
    mGpuThreadBlockSize = std::clamp(blockSize, GpuIndex{32}, GpuIndex{256});
}

common::Buffer<GpuScalar, 3> const& Integrator::GetVelocity() const
{
    return mVelocities;
}

common::Buffer<GpuScalar, 3> const& Integrator::GetExternalAcceleration() const
{
    return mExternalAcceleration;
}

common::Buffer<GpuScalar> const& Integrator::GetMass() const
{
    return mMass;
}

common::Buffer<GpuScalar> const& Integrator::GetShapeFunctionGradients() const
{
    return mShapeFunctionGradients;
}

common::Buffer<GpuScalar> const& Integrator::GetLameCoefficients() const
{
    return mLameCoefficients;
}

} // namespace vbd
} // namespace impl
} // namespace gpu
} // namespace pbat

#include "pbat/common/Eigen.h"

#include <Eigen/SparseCore>
#include <doctest/doctest.h>
#include <span>
#include <vector>

TEST_CASE("[gpu][impl][vbd] Integrator")
{
    using namespace pbat;
    using pbat::common::ToEigen;
    // Arrange
    // Cube mesh
    MatrixX P(3, 8);
    IndexMatrixX V(1, 8);
    IndexMatrixX T(4, 5);
    IndexMatrixX F(3, 12);
    // clang-format off
    P << 0.f, 1.f, 0.f, 1.f, 0.f, 1.f, 0.f, 1.f,
         0.f, 0.f, 1.f, 1.f, 0.f, 0.f, 1.f, 1.f,
         0.f, 0.f, 0.f, 0.f, 1.f, 1.f, 1.f, 1.f;
    T << 0, 3, 5, 6, 0,
         1, 2, 4, 7, 5,
         3, 0, 6, 5, 3,
         5, 6, 0, 3, 6;
    F << 0, 1, 1, 3, 3, 2, 2, 0, 0, 0, 4, 5,
         1, 5, 3, 7, 2, 6, 0, 4, 3, 2, 5, 7,
         4, 4, 5, 5, 7, 7, 6, 6, 1, 3, 6, 6;
    // clang-format on
    V.reshaped().setLinSpaced(0, static_cast<Index>(P.cols() - 1));
    // Problem parameters
    auto constexpr dt         = GpuScalar{1e-2};
    auto constexpr substeps   = 1;
    auto constexpr iterations = 10;

    // Act
    using pbat::gpu::impl::vbd::Integrator;
    Integrator vbd{sim::vbd::Data().WithVolumeMesh(P, T).WithSurfaceMesh(V, F).Construct()};
    vbd.Step(dt, iterations, substeps);

    // Assert
    auto constexpr zero = GpuScalar{1e-4};
    GpuMatrixX dx =
        ToEigen(vbd.X.x.Get()).reshaped(P.cols(), P.rows()).transpose() - P.cast<GpuScalar>();
    bool const bVerticesFallUnderGravity = (dx.row(2).array() < GpuScalar{0}).all();
    CHECK(bVerticesFallUnderGravity);
    bool const bVerticesOnlyFall = (dx.topRows(2).array().abs() < zero).all();
    CHECK(bVerticesOnlyFall);
}
