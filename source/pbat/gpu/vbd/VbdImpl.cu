#include "hip/hip_runtime.h"
// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "VbdImpl.cuh"
#include "VbdImplKernels.cuh"
#include "pbat/gpu/common/Cuda.cuh"

#include <cuda/api.hpp>
// #include <thrust/async/copy.h>
#include <thrust/async/for_each.h>
#include <thrust/execution_policy.h>

namespace pbat {
namespace gpu {
namespace vbd {

VbdImpl::VbdImpl(
    Eigen::Ref<GpuMatrixX const> const& Xin,
    Eigen::Ref<GpuIndexMatrixX const> const& Vin,
    Eigen::Ref<GpuIndexMatrixX const> const& Fin,
    Eigen::Ref<GpuIndexMatrixX const> const& Tin)
    : X(Xin),
      V(Vin),
      F(Fin),
      T(Tin),
      mPositionsAtT(Xin.cols()),
      mKineticEnergyMinimalPositions(Xin.cols()),
      mChebyshevPositionsM2(Xin.cols()),
      mChebyshevPositionsM1(Xin.cols()),
      mVelocitiesAtT(Xin.cols()),
      mVelocities(Xin.cols()),
      mExternalAcceleration(Xin.cols()),
      mMass(Xin.cols()),
      mQuadratureWeights(Tin.cols()),
      mShapeFunctionGradients(Tin.cols() * 4 * 3),
      mLameCoefficients(2 * Tin.cols()),
      mDetHZero(GpuScalar{1e-10}),
      mVertexTetrahedronPrefix(Xin.cols() + 1),
      mVertexTetrahedronNeighbours(),
      mVertexTetrahedronLocalVertexIndices(),
      mRayleighDamping(GpuScalar{0}),
      mCollisionPenalty(GpuScalar{1e3}),
      mMaxCollidingTrianglesPerVertex(8),
      mCollidingTriangles(8 * Xin.cols()),
      mCollidingTriangleCount(Xin.cols()),
      mPartitions(),
      mInitializationStrategy(EInitializationStrategy::AdaptiveVbd),
      mGpuThreadBlockSize(64),
      mStream(common::Device(common::EDeviceSelectionPreference::HighestComputeCapability)
                  .create_stream(/*synchronize_with_default_stream=*/false))
{
    for (auto d = 0; d < X.Dimensions(); ++d)
    {
        thrust::copy(X.x[d].begin(), X.x[d].end(), mPositionsAtT[d].begin());
        thrust::fill(mVelocitiesAtT[d].begin(), mVelocitiesAtT[d].end(), GpuScalar{0});
        thrust::fill(mVelocities[d].begin(), mVelocities[d].end(), GpuScalar{0});
        thrust::fill(
            mExternalAcceleration[d].begin(),
            mExternalAcceleration[d].end(),
            GpuScalar{0});
    }
    thrust::fill(mMass.Data(), mMass.Data() + mMass.Size(), GpuScalar{1e3});
}

void VbdImpl::Step(GpuScalar dt, GpuIndex iterations, GpuIndex substeps, GpuScalar rho)
{
    GpuScalar sdt                        = dt / static_cast<GpuScalar>(substeps);
    GpuScalar sdt2                       = sdt * sdt;
    GpuIndex const nVertices             = static_cast<GpuIndex>(X.NumberOfPoints());
    bool const bUseChebyshevAcceleration = rho > GpuScalar{0} and rho < GpuScalar{1};

    kernels::BackwardEulerMinimization bdf{};
    bdf.dt                              = sdt;
    bdf.dt2                             = sdt2;
    bdf.m                               = mMass.Raw();
    bdf.xtilde                          = mKineticEnergyMinimalPositions.Raw();
    bdf.xt                              = mPositionsAtT.Raw();
    bdf.x                               = X.x.Raw();
    bdf.T                               = T.inds.Raw();
    bdf.wg                              = mQuadratureWeights.Raw();
    bdf.GP                              = mShapeFunctionGradients.Raw();
    bdf.lame                            = mLameCoefficients.Raw();
    bdf.detHZero                        = mDetHZero;
    bdf.GVTp                            = mVertexTetrahedronPrefix.Raw();
    bdf.GVTn                            = mVertexTetrahedronNeighbours.Raw();
    bdf.GVTilocal                       = mVertexTetrahedronLocalVertexIndices.Raw();
    bdf.kD                              = mRayleighDamping;
    bdf.kC                              = mCollisionPenalty;
    bdf.nMaxCollidingTrianglesPerVertex = mMaxCollidingTrianglesPerVertex;
    bdf.FC                              = mCollidingTriangles.Raw();
    bdf.nCollidingTriangles             = mCollidingTriangleCount.Raw();
    bdf.F                               = F.inds.Raw();

    // NOTE:
    // For some reason, thrust::async::copy does not play well with cuda-api-wrapper streams. I am
    // guessing it has to do with synchronize_with_default_stream=false?
    mStream.device().make_current();
    for (auto s = 0; s < substeps; ++s)
    {
        // Store previous positions
        for (auto d = 0; d < X.x.Dimensions(); ++d)
        {
            cuda::memory::async::copy(
                thrust::raw_pointer_cast(mPositionsAtT[d].data()),
                thrust::raw_pointer_cast(X.x[d].data()),
                X.x.Size() * sizeof(GpuScalar),
                mStream);
        }
        // Compute inertial target positions
        thrust::device_event e = thrust::async::for_each(
            // Share thrust's underlying CUDA stream with cuda-api-wrappers
            thrust::device.on(mStream.handle()),
            thrust::make_counting_iterator<GpuIndex>(0),
            thrust::make_counting_iterator<GpuIndex>(nVertices),
            kernels::FKineticEnergyMinimum{
                sdt,
                sdt2,
                X.x.Raw(),
                mVelocities.Raw(),
                mExternalAcceleration.Raw(),
                mKineticEnergyMinimalPositions.Raw()});
        // Initialize block coordinate descent's, i.e. BCD's, solution
        e = thrust::async::for_each(
            thrust::device.on(mStream.handle()),
            thrust::make_counting_iterator<GpuIndex>(0),
            thrust::make_counting_iterator<GpuIndex>(nVertices),
            kernels::FAdaptiveInitialization{
                sdt,
                sdt2,
                mPositionsAtT.Raw(),
                mVelocitiesAtT.Raw(),
                mVelocities.Raw(),
                mExternalAcceleration.Raw(),
                X.x.Raw(),
                mInitializationStrategy});
        // Initialize Chebyshev semi-iterative method
        kernels::FChebyshev fChebyshev{
            rho,
            mChebyshevPositionsM2.Raw(),
            mChebyshevPositionsM1.Raw(),
            X.x.Raw()};
        auto kDynamicSharedMemoryCapacity = static_cast<cuda::memory::shared::size_t>(
            mGpuThreadBlockSize * bdf.ExpectedSharedMemoryPerThreadInBytes());
        // Minimize Backward Euler, i.e. BDF1, objective
        for (auto k = 0; k < iterations; ++k)
        {
            if (bUseChebyshevAcceleration)
                fChebyshev.SetIteration(k);

            for (auto& partition : mPartitions)
            {
                bdf.partition = partition.Raw();
                auto const nVerticesInPartition =
                    static_cast<cuda::grid::dimension_t>(partition.Size());
                auto bcdLaunchConfiguration =
                    cuda::launch_config_builder()
                        .block_size(mGpuThreadBlockSize)
                        .dynamic_shared_memory_size(kDynamicSharedMemoryCapacity)
                        .grid_size(nVerticesInPartition)
                        .build();
                mStream.enqueue.kernel_launch(
                    kernels::MinimizeBackwardEuler,
                    bcdLaunchConfiguration,
                    bdf);
            }

            if (bUseChebyshevAcceleration)
            {
                e = thrust::async::for_each(
                    thrust::device.on(mStream.handle()),
                    thrust::make_counting_iterator<GpuIndex>(0),
                    thrust::make_counting_iterator<GpuIndex>(nVertices),
                    fChebyshev);
            }
        }
        // Update velocities
        for (auto d = 0; d < mVelocities.Dimensions(); ++d)
        {
            cuda::memory::async::copy(
                thrust::raw_pointer_cast(mVelocitiesAtT[d].data()),
                thrust::raw_pointer_cast(mVelocities[d].data()),
                mVelocities.Size() * sizeof(GpuScalar),
                mStream);
        }
        e = thrust::async::for_each(
            thrust::device.on(mStream.handle()),
            thrust::make_counting_iterator<GpuIndex>(0),
            thrust::make_counting_iterator<GpuIndex>(nVertices),
            kernels::FUpdateVelocity{sdt, mPositionsAtT.Raw(), X.x.Raw(), mVelocities.Raw()});
    }
    mStream.synchronize();
}

void VbdImpl::SetPositions(Eigen::Ref<GpuMatrixX const> const& Xin)
{
    auto const nVertices = static_cast<GpuIndex>(X.x.Size());
    if (Xin.rows() != 3 and Xin.cols() != nVertices)
    {
        std::ostringstream ss{};
        ss << "Expected positions of dimensions " << X.x.Dimensions() << "x" << X.x.Size()
           << ", but got " << Xin.rows() << "x" << Xin.cols() << "\n";
        throw std::invalid_argument(ss.str());
    }
    for (auto d = 0; d < X.x.Dimensions(); ++d)
    {
        thrust::copy(Xin.row(d).begin(), Xin.row(d).end(), X.x[d].begin());
    }
}

void VbdImpl::SetVelocities(Eigen::Ref<GpuMatrixX const> const& v)
{
    auto const nVertices = static_cast<GpuIndex>(mVelocities.Size());
    if (v.rows() != 3 and v.cols() != nVertices)
    {
        std::ostringstream ss{};
        ss << "Expected velocities of dimensions " << mVelocities.Dimensions() << "x"
           << mVelocities.Size() << ", but got " << v.rows() << "x" << v.cols() << "\n";
        throw std::invalid_argument(ss.str());
    }
    for (auto d = 0; d < mVelocities.Dimensions(); ++d)
    {
        thrust::copy(v.row(d).begin(), v.row(d).end(), mVelocities[d].begin());
    }
}

void VbdImpl::SetExternalAcceleration(Eigen::Ref<GpuMatrixX const> const& aext)
{
    auto const nVertices = static_cast<GpuIndex>(mExternalAcceleration.Size());
    if (aext.rows() != 3 and aext.cols() != nVertices)
    {
        std::ostringstream ss{};
        ss << "Expected accelerations of dimensions " << mExternalAcceleration.Dimensions() << "x"
           << mExternalAcceleration.Size() << ", but got " << aext.rows() << "x" << aext.cols()
           << "\n";
        throw std::invalid_argument(ss.str());
    }
    for (auto d = 0; d < mExternalAcceleration.Dimensions(); ++d)
        thrust::copy(aext.row(d).begin(), aext.row(d).end(), mExternalAcceleration[d].begin());
}

void VbdImpl::SetMass(Eigen::Ref<GpuVectorX const> const& m)
{
    auto const nVertices = static_cast<GpuIndex>(mMass.Size());
    if (m.size() != nVertices)
    {
        std::ostringstream ss{};
        ss << "Expected masses of dimensions " << nVertices << "x1 or its transpose, but got "
           << m.size() << "\n";
        throw std::invalid_argument(ss.str());
    }
    thrust::copy(m.data(), m.data() + m.size(), mMass.Data());
}

void VbdImpl::SetQuadratureWeights(Eigen::Ref<GpuVectorX const> const& wg)
{
    auto const nTetrahedra = static_cast<GpuIndex>(T.inds.Size());
    if (wg.size() != nTetrahedra)
    {
        std::ostringstream ss{};
        ss << "Expected quadrature weights of dimensions " << nTetrahedra
           << "x1 or its transpose, but got " << wg.rows() << "x" << wg.cols() << "\n";
        throw std::invalid_argument(ss.str());
    }
    thrust::copy(wg.data(), wg.data() + wg.size(), mQuadratureWeights.Data());
}

void VbdImpl::SetShapeFunctionGradients(Eigen::Ref<GpuMatrixX const> const& GP)
{
    auto const nTetrahedra = static_cast<GpuIndex>(T.inds.Size());
    if (GP.rows() != 4 and GP.cols() != nTetrahedra * 3)
    {
        std::ostringstream ss{};
        ss << "Expected shape function gradients of dimensions 4x" << nTetrahedra * 3
           << ", but got " << GP.rows() << "x" << GP.cols() << "\n";
        throw std::invalid_argument(ss.str());
    }
    thrust::copy(GP.data(), GP.data() + GP.size(), mShapeFunctionGradients.Data());
}

void VbdImpl::SetLameCoefficients(Eigen::Ref<GpuMatrixX const> const& l)
{
    auto const nTetrahedra = static_cast<GpuIndex>(T.inds.Size());
    if (l.rows() != 2 and l.cols() != nTetrahedra)
    {
        std::ostringstream ss{};
        ss << "Expected Lame coefficients of dimensions 2x" << nTetrahedra << ", but got "
           << l.rows() << "x" << l.cols() << "\n";
        throw std::invalid_argument(ss.str());
    }
    thrust::copy(l.data(), l.data() + l.size(), mLameCoefficients.Data());
}

void VbdImpl::SetNumericalZeroForHessianDeterminant(GpuScalar zero)
{
    mDetHZero = zero;
}

void VbdImpl::SetVertexTetrahedronAdjacencyList(
    Eigen::Ref<GpuIndexVectorX const> const& GVTp,
    Eigen::Ref<GpuIndexVectorX const> const& GVTn,
    Eigen::Ref<GpuIndexVectorX const> const& GVTilocal)
{
    if (static_cast<std::size_t>(GVTp.size()) != mVertexTetrahedronPrefix.Size())
    {
        std::ostringstream ss{};
        ss << "Expected vertex-tetrahedron adjacency graph's prefix array to have size="
           << mVertexTetrahedronPrefix.Size() << ", but got " << GVTp.size() << "\n";
        throw std::invalid_argument(ss.str());
    }
    if (GVTn.size() != GVTilocal.size())
    {
        std::ostringstream ss{};
        ss << "Expected vertex-tetrahedron adjacency graph's neighbour array and data (ilocal) "
              "array to have the same size, but got neighbours="
           << GVTn.size() << ", ilocal=" << GVTilocal.size() << " \n";
        throw std::invalid_argument(ss.str());
    }

    mVertexTetrahedronNeighbours.Resize(GVTn.size());
    mVertexTetrahedronLocalVertexIndices.Resize(GVTilocal.size());

    thrust::copy(GVTp.data(), GVTp.data() + GVTp.size(), mVertexTetrahedronPrefix.Data());
    thrust::copy(GVTn.data(), GVTn.data() + GVTn.size(), mVertexTetrahedronNeighbours.Data());
    thrust::copy(
        GVTilocal.data(),
        GVTilocal.data() + GVTilocal.size(),
        mVertexTetrahedronLocalVertexIndices.Data());
}

void VbdImpl::SetRayleighDampingCoefficient(GpuScalar kD)
{
    mRayleighDamping = kD;
}

void VbdImpl::SetVertexPartitions(std::vector<std::vector<GpuIndex>> const& partitions)
{
    mPartitions.resize(partitions.size());
    for (auto p = 0; p < partitions.size(); ++p)
    {
        mPartitions[p].Resize(partitions[p].size());
        thrust::copy(partitions[p].begin(), partitions[p].end(), mPartitions[p].Data());
    }
}

void VbdImpl::SetInitializationStrategy(EInitializationStrategy strategy)
{
    mInitializationStrategy = strategy;
}

void VbdImpl::SetBlockSize(GpuIndex blockSize)
{
    mGpuThreadBlockSize = blockSize;
}

common::Buffer<GpuScalar, 3> const& VbdImpl::GetVelocity() const
{
    return mVelocities;
}

common::Buffer<GpuScalar, 3> const& VbdImpl::GetExternalAcceleration() const
{
    return mExternalAcceleration;
}

common::Buffer<GpuScalar> const& VbdImpl::GetMass() const
{
    return mMass;
}

common::Buffer<GpuScalar> const& VbdImpl::GetShapeFunctionGradients() const
{
    return mShapeFunctionGradients;
}

common::Buffer<GpuScalar> const& VbdImpl::GetLameCoefficients() const
{
    return mLameCoefficients;
}

std::vector<common::Buffer<GpuIndex>> const& VbdImpl::GetPartitions() const
{
    return mPartitions;
}

} // namespace vbd
} // namespace gpu
} // namespace pbat

#include "pbat/common/Eigen.h"
#include "tests/Fem.h"

#include <Eigen/SparseCore>
#include <doctest/doctest.h>
#include <span>
#include <vector>

TEST_CASE("[gpu][xpbd] Xpbd")
{
    using pbat::GpuIndex;
    using pbat::GpuIndexMatrixX;
    using pbat::GpuMatrixX;
    using pbat::GpuScalar;
    using pbat::GpuVectorX;
    using pbat::Index;
    using pbat::Scalar;
    using pbat::common::ToEigen;
    // Arrange
    // Cube mesh
    GpuMatrixX P(3, 8);
    GpuIndexMatrixX V(1, 8);
    GpuIndexMatrixX T(4, 5);
    GpuIndexMatrixX F(3, 12);
    // clang-format off
    P << 0.f, 1.f, 0.f, 1.f, 0.f, 1.f, 0.f, 1.f,
         0.f, 0.f, 1.f, 1.f, 0.f, 0.f, 1.f, 1.f,
         0.f, 0.f, 0.f, 0.f, 1.f, 1.f, 1.f, 1.f;
    T << 0, 3, 5, 6, 0,
         1, 2, 4, 7, 5,
         3, 0, 6, 5, 3,
         5, 6, 0, 3, 6;
    F << 0, 1, 1, 3, 3, 2, 2, 0, 0, 0, 4, 5,
         1, 5, 3, 7, 2, 6, 0, 4, 3, 2, 5, 7,
         4, 4, 5, 5, 7, 7, 6, 6, 1, 3, 6, 6;
    // clang-format on
    V.reshaped().setLinSpaced(0, static_cast<GpuIndex>(P.cols() - 1));
    // Parallel graph information
    using SparseMatrixType = Eigen::SparseMatrix<GpuIndex, Eigen::ColMajor>;
    using TripletType      = Eigen::Triplet<GpuIndex, typename SparseMatrixType::StorageIndex>;
    SparseMatrixType G(T.cols(), P.cols());
    std::vector<TripletType> Gei{};
    for (auto e = 0; e < T.cols(); ++e)
    {
        for (auto ilocal = 0; ilocal < T.rows(); ++ilocal)
        {
            auto i = T(ilocal, e);
            Gei.push_back(TripletType{e, i, ilocal});
        }
    }
    G.setFromTriplets(Gei.begin(), Gei.end());
    assert(G.isCompressed());
    std::span<GpuIndex> vertexTetrahedronPrefix{
        G.outerIndexPtr(),
        static_cast<std::size_t>(G.outerSize() + 1)};
    std::span<GpuIndex> vertexTetrahedronNeighbours{
        G.innerIndexPtr(),
        static_cast<std::size_t>(G.nonZeros())};
    std::span<GpuIndex> vertexTetrahedronLocalVertexIndices{
        G.valuePtr(),
        static_cast<std::size_t>(G.nonZeros())};
    std::vector<std::vector<GpuIndex>> partitions{};
    partitions.push_back({2, 7, 4, 1});
    partitions.push_back({0});
    partitions.push_back({5});
    partitions.push_back({6});
    partitions.push_back({3});
    // Material parameters
    using pbat::gpu::vbd::tests::LinearFemMesh;
    LinearFemMesh mesh{P, T};
    GpuVectorX wg     = mesh.QuadratureWeights();
    GpuMatrixX GP     = mesh.ShapeFunctionGradients();
    auto constexpr Y  = GpuScalar{1e6};
    auto constexpr nu = GpuScalar{0.45};
    GpuMatrixX lame   = mesh.LameCoefficients(Y, nu);
    // Problem parameters
    GpuMatrixX aext(3, P.cols());
    aext.colwise()    = Eigen::Vector<GpuScalar, 3>{GpuScalar{0}, GpuScalar{0}, GpuScalar{-9.81}};
    auto constexpr dt = GpuScalar{1e-2};
    auto constexpr substeps   = 1;
    auto constexpr iterations = 10;

    // Act
    using pbat::gpu::vbd::VbdImpl;
    VbdImpl vbd{P, V, F, T};
    vbd.SetExternalAcceleration(aext);
    vbd.SetQuadratureWeights(wg);
    vbd.SetShapeFunctionGradients(GP);
    vbd.SetLameCoefficients(lame);
    vbd.SetVertexTetrahedronAdjacencyList(
        ToEigen(vertexTetrahedronPrefix),
        ToEigen(vertexTetrahedronNeighbours),
        ToEigen(vertexTetrahedronLocalVertexIndices));
    vbd.SetVertexPartitions(partitions);
    vbd.Step(dt, iterations, substeps);

    // Assert
    auto constexpr zero = GpuScalar{1e-4};
    GpuMatrixX dx       = ToEigen(vbd.X.x.Get()).reshaped(P.cols(), P.rows()).transpose() - P;
    bool const bVerticesFallUnderGravity = (dx.row(2).array() < GpuScalar{0}).all();
    CHECK(bVerticesFallUnderGravity);
    bool const bVerticesOnlyFall = (dx.topRows(2).array().abs() < zero).all();
    CHECK(bVerticesOnlyFall);
}
